#include "hip/hip_runtime.h"
#include "kangaroo.h"
#include "launch_utils.h"

namespace Gpu {

template<typename TO, typename TI>
__global__ void KernBlurX(Image<TO> out, Image<TI> in)
{
    const unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x==0) {
        out(x,y) = (2*in(x,y) + in(x+1,y)) / 3.0f;
    }else if(x== in.w-1) {
        out(x,y) = (2*in(x,y) + in(x-1,y)) / 3.0f;
    }else{
        out(x,y) = (in(x-1,y) + 2*in(x,y) + in(x+1,y)) / 4.0f;
    }
}

template<typename TO, typename TI>
__global__ void KernBlurY(Image<TO> out, Image<TI> in)
{
    const unsigned x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y*blockDim.y + threadIdx.y;

    if(y==0) {
        out(x,y) = (2*in(x,y) + in(x,y+1)) / 3.0f;
    }else if(y== in.h-1) {
        out(x,y) = (2*in(x,y) + in(x,y-1)) / 3.0f;
    }else{
        out(x,y) = (in(x,y-1) + 2*in(x,y) + in(x,y+1)) / 4.0f;
    }
}

void Blur(Image<unsigned char> out, Image<unsigned char> in, Image<unsigned char> temp )
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, out, 16, 16);
    KernBlurX<unsigned char,unsigned char><<<gridDim,blockDim>>>(temp,in);
    KernBlurY<unsigned char,unsigned char><<<gridDim,blockDim>>>(out,temp);
}

}
