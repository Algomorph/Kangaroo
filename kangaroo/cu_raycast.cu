#include "hip/hip_runtime.h"
#include "Mat.h"
#include "MatUtils.h"
#include "Image.h"
#include "Volume.h"
#include "Sdf.h"
#include "launch_utils.h"

namespace Gpu
{

__global__ void KernRaycastSDF(Image<float> img, const Volume<SDF_t> vol, const float3 boxmin, const float3 boxmax, const Mat<float,3,4> T_wc, float fu, float fv, float u0, float v0, float near, float far, bool subpix )
{
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;

    if( u < img.w && v < img.h ) {
        const float3 c_w = SE3Translation(T_wc);
        const float3 ray_c = make_float3((u-u0)/fu,(v-v0)/fv, 1);
        const float3 ray_w = mulSO3(T_wc, ray_c);

        // Raycast bounding box to find valid ray segment of sdf
        // http://www.cs.utah.edu/~awilliam/box/box.pdf
        const float3 tminbound = (boxmin - c_w) / ray_w;
        const float3 tmaxbound = (boxmax - c_w) / ray_w;
        const float3 tmin = fminf(tminbound,tmaxbound);
        const float3 tmax = fmaxf(tminbound,tmaxbound);
        const float max_tmin = fmaxf(fmaxf(fmaxf(tmin.x, tmin.y), tmin.z), near);
        const float min_tmax = fminf(fminf(fminf(tmax.x, tmax.y), tmax.z), far);

        float ret = 0.0f;

        if(max_tmin < min_tmax ) {
            // Go between max_tmin and min_tmax
            float lambda = max_tmin;
            float last_sdf = 0;
            float delta_lambda = (boxmax.x - boxmin.x) / (vol.w-1);

            while(lambda < min_tmax) {
                const float3 pos_w = c_w + lambda * ray_w;
                const float3 pos_v = (pos_w - boxmin) / (boxmax - boxmin);
                const float sdf = vol.GetFractionalTrilinearClamped(pos_v);
//                const float sdf = val.val / val.n;
                if( sdf <= 0 ) {
                    // surface!
                    if(subpix) {
                        lambda = lambda - delta_lambda * last_sdf / (sdf - last_sdf);
                    }
                    ret = (lambda - near) / (far - near);
                    break;
                }
                lambda += fmaxf(delta_lambda, sdf);
                last_sdf = sdf;
            }
        }

        img(u,v) = ret;
    }
}

void Raycast(Image<float> img, const Volume<SDF_t> vol, const float3 boxmin, const float3 boxmax, const Mat<float,3,4> T_wc, float fu, float fv, float u0, float v0, float near, float far, bool subpix )
{    
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim, gridDim, img);
    KernRaycastSDF<<<gridDim,blockDim>>>(img, vol, boxmin, boxmax, T_wc, fu, fv, u0, v0, near, far, subpix);
}

__global__ void KernSDFSphere(Volume<SDF_t> vol, float3 vol_min, float3 vol_max, float3 center, float r)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int z = blockIdx.z*blockDim.z + threadIdx.z;

    const float3 vol_size = vol_max - vol_min;

    const float3 pos = make_float3(
                vol_min.x + vol_size.x*x/(float)(vol.w-1),
                vol_min.y + vol_size.y*y/(float)(vol.h-1),
                vol_min.z + vol_size.z*z/(float)(vol.d-1)
                );
    const float dist = length(pos - center);
    const float sdf = dist - r;

    vol(x,y,z) = SDF_t(sdf);
}

void SDFSphere(Volume<SDF_t> vol, float3 vol_min, float3 vol_max, float3 center, float r)
{
    dim3 blockDim(8,8,8);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);

    KernSDFSphere<<<gridDim,blockDim>>>(vol, vol_min, vol_max, center, r);
}


}
