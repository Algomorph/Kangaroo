#include "hip/hip_runtime.h"
#include "MatUtils.h"
#include "Image.h"
#include "Sdf.h"
#include "BoundedVolume.h"
#include "launch_utils.h"
#include "ImageKeyframe.h"

namespace Gpu
{

//////////////////////////////////////////////////////
// Truncated SDF Fusion
//////////////////////////////////////////////////////

__global__ void KernSdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K, float trunc_dist, float max_w, float mincostheta )
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int z = blockIdx.z*blockDim.z + threadIdx.z;

    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    const float3 P_c = T_cw * P_w;
    const float2 p_c = K.Project(P_c);

    if( depth.InBounds(p_c, 2) )
    {
        const float vd = P_c.z;
//        const float md = depth.GetNearestNeighbour(p_c);
//        const float3 mdn = make_float3(normals.GetNearestNeighbour(p_c));

        const float md = depth.GetBilinear<float>(p_c);
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist) {
            // Further than truncation distance from surface
            // We do nothing.
        }else{
//        }else if(sd < 5*trunc_dist) {
            if(isfinite(md) && isfinite(w) && costheta > mincostheta ) {
                SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
                sdf += vol(x,y,z);
//                sdf.Clamp(-trunc_dist, trunc_dist);
                sdf.LimitWeight(max_w);
                vol(x,y,z) = sdf;
            }
        }
    }
 }

using namespace std;
void SdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K, float trunc_dist, float max_w, float mincostheta )
{
    dim3 blockDim(8,8,8);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);
    KernSdfFuse<<<gridDim,blockDim>>>(vol, depth, norm, T_cw, K, trunc_dist, max_w, mincostheta);
    GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Reset SDF
//////////////////////////////////////////////////////

__global__ void KernSdfReset(BoundedVolume<SDF_t> vol, float trunc_dist)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int z = blockIdx.z*blockDim.z + threadIdx.z;

    vol(x,y,z) = SDF_t(0.0/0.0, 0);
}

void SdfReset(BoundedVolume<SDF_t> vol, float trunc_dist)
{
    dim3 blockDim(8,8,8);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);
    KernSdfReset<<<gridDim,blockDim>>>(vol, trunc_dist);
    GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Create SDF representation of sphere
//////////////////////////////////////////////////////

__global__ void KernSdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int z = blockIdx.z*blockDim.z + threadIdx.z;

    const float3 pos = vol.VoxelPositionInUnits(x,y,z);
    const float dist = length(pos - center);
    const float sdf = dist - r;

    vol(x,y,z) = SDF_t(sdf);
}

void SdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
    dim3 blockDim(8,8,8);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);

    KernSdfSphere<<<gridDim,blockDim>>>(vol, center, r);
    GpuCheckErrors();
}


}
