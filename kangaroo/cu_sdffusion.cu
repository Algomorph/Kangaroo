#include "hip/hip_runtime.h"
#include "MatUtils.h"
#include "Image.h"
#include "Sdf.h"
#include "BoundedVolume.h"
#include "launch_utils.h"

namespace Gpu
{

//////////////////////////////////////////////////////
// Truncated SDF Fusion
//////////////////////////////////////////////////////

__global__ void KernSdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, float fu, float fv, float u0, float v0, float trunc_dist, float max_w, float mincostheta )
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int z = blockIdx.z*blockDim.z + threadIdx.z;

    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    const float3 P_c = T_cw * P_w;
    const float2 p_c = make_float2(u0 + fu*P_c.x/P_c.z, v0 + fv*P_c.y/P_c.z);

    if( depth.InBounds(p_c, 2) )
    {
        const float vd = P_c.z;
//        const float md = depth.GetNearestNeighbour(p_c);
//        const float3 mdn = make_float3(normals.GetNearestNeighbour(p_c));

        const float md = depth.GetBilinear<float>(p_c);
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist) {
            // Further than truncation distance from surface
            // We do nothing.
        }else{
//        }else if(sd < 5*trunc_dist) {
            if(isfinite(md) && isfinite(w) && costheta > mincostheta ) {
                SDF_t sdf = SDF_t(sd, w) + vol(x,y,z);
                sdf.Clamp(-trunc_dist, trunc_dist);
                sdf.LimitWeight(max_w);
                vol(x,y,z) = sdf;
            }
        }
    }
 }

using namespace std;
void SdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, float fu, float fv, float u0, float v0, float trunc_dist, float max_w, float mincostheta )
{
    dim3 blockDim(8,8,8);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);
    KernSdfFuse<<<gridDim,blockDim>>>(vol, depth, norm, T_cw, fu, fv, u0, v0, trunc_dist, max_w, mincostheta);
    GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Reset SDF
//////////////////////////////////////////////////////

__global__ void KernSdfReset(BoundedVolume<SDF_t> vol, float trunc_dist)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int z = blockIdx.z*blockDim.z + threadIdx.z;

    vol(x,y,z) = SDF_t(trunc_dist, 0);
}

void SdfReset(BoundedVolume<SDF_t> vol, float trunc_dist)
{
    dim3 blockDim(8,8,8);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);
    KernSdfReset<<<gridDim,blockDim>>>(vol, trunc_dist);
    GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Create SDF representation of sphere
//////////////////////////////////////////////////////

__global__ void KernSdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int z = blockIdx.z*blockDim.z + threadIdx.z;

    const float3 pos = vol.VoxelPositionInUnits(x,y,z);
    const float dist = length(pos - center);
    const float sdf = dist - r;

    vol(x,y,z) = SDF_t(sdf);
}

void SdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
    dim3 blockDim(8,8,8);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);

    KernSdfSphere<<<gridDim,blockDim>>>(vol, center, r);
    GpuCheckErrors();
}


}
