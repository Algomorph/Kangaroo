#include "hip/hip_runtime.h"
#include "all.h"
#include "launch_utils.h"

namespace Gpu
{

//////////////////////////////////////////////////////
// Make Index Buffer for rendering
//////////////////////////////////////////////////////

__global__ void KernGenerateTriangleStripIndexBuffer(Image<uint2> dIbo)
{
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    const unsigned int pixIndex = y*dIbo.w + x;
    dIbo(x,y) = make_uint2(pixIndex, pixIndex + dIbo.w);
}

void GenerateTriangleStripIndexBuffer( Image<uint2> dIbo)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dIbo);
    KernGenerateTriangleStripIndexBuffer<<<gridDim,blockDim>>>(dIbo);
}

}
