#include "hip/hip_runtime.h"
#include "all.h"
#include "launch_utils.h"

namespace Gpu
{

//////////////////////////////////////////////////////
// Plane Fitting
//////////////////////////////////////////////////////

void InitHeightMap(Image<float4> dHeightMap)
{
    // initialize the heightmap
    dHeightMap.Fill(make_float4(0,10000.0,128,0.0));
}

//////////////////////////////////////////////////////

__global__ void KernUpdateHeightmap(Image<float4> dHeightMap, const Image<float4> d3d, const Image<unsigned char> dImage,  const Mat<float,3,4> T_hc)
{
    const unsigned int u = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int v = blockIdx.y*blockDim.y + threadIdx.y;

    //calculate the position in heightmap coordinates
    float4 p_c = d3d(u,v);
    float3 p_h = make_float3(T_hc(0,0)*p_c.x + T_hc(0,1)*p_c.y + T_hc(0,2)*p_c.z + T_hc(0,3)*p_c.w,
                             T_hc(1,0)*p_c.x + T_hc(1,1)*p_c.y + T_hc(1,2)*p_c.z + T_hc(1,3)*p_c.w,
                             T_hc(2,0)*p_c.x + T_hc(2,1)*p_c.y + T_hc(2,2)*p_c.z + T_hc(2,3)*p_c.w);

    int x = (int)(p_h.x+0.5);
    int y = (int)(p_h.y+0.5);

    if(dHeightMap.InBounds(x,y) && isfinite(p_h.z) ) {
        //calculate the variance of the measurement
        float v_z = p_c.z*0.01; //this is the perp. distance from the camera
        unsigned char colour = dImage.IsValid() ? dImage(u,v) : 0;
        float4 oldVal = dHeightMap(x,y);
        float4 newVal = make_float4((oldVal.y * p_h.z + v_z * oldVal.x)/(oldVal.y+v_z),
                                    oldVal.y*v_z / (oldVal.y+v_z),
                                    (oldVal.y * colour + v_z * oldVal.z)/(oldVal.y+v_z),
                                    0.0);

        // Take new val
//        float4 newVal = make_float4(p_h.z, 0, dImage(u,v), 0);

        dHeightMap(x,y) = newVal;
    }
}

void UpdateHeightMap(Image<float4> dHeightMap, const Image<float4> d3d, const Image<unsigned char> dImage, const Mat<float,3,4> T_hc)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, d3d);
    KernUpdateHeightmap<<<gridDim,blockDim>>>(dHeightMap,d3d,dImage,T_hc);
}

//////////////////////////////////////////////////////

__global__ void KernVboFromHeightmap(Image<float4> dVbo, const Image<float4> dHeightMap)
{
    const unsigned int u = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int v = blockIdx.y*blockDim.y + threadIdx.y;

    dVbo(u,v) = make_float4(u,v,dHeightMap(u,v).x,1.0);
}


void VboFromHeightMap(Image<float4> dVbo, const Image<float4> dHeightMap)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dVbo);
    KernVboFromHeightmap<<<gridDim,blockDim>>>(dVbo,dHeightMap);
}

//////////////////////////////////////////////////////

__global__ void KernColourHeightmap(Image<uchar4> dCbo, const Image<float4> dHeightMap)
{
    const unsigned int u = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int v = blockIdx.y*blockDim.y + threadIdx.y;

    float v_z = dHeightMap(u,v).z;
//    dCbo(u,v) = make_uchar4(255,0,0,255);
    dCbo(u,v) = make_uchar4(v_z,v_z,v_z,255);
}

void ColourHeightMap(Image<uchar4> dCbo, const Image<float4> dHeightMap)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dCbo);
    KernColourHeightmap<<<gridDim,blockDim>>>(dCbo,dHeightMap);
}

//////////////////////////////////////////////////////

__global__ void KernGenerateHeightAndImageFromHeightmap(Image<float> dHeight, Image<unsigned char> dImage, const Image<float4> dHeightMap)
{
    const unsigned int u = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int v = blockIdx.y*blockDim.y + threadIdx.y;

    dHeight(u,v) = dHeightMap(u,v).x;
    dImage(u,v) = dHeightMap(u,v).z;
}

void GenerateHeightAndImageFromHeightmap(Image<float> dHeight, Image<unsigned char> dImage, const Image<float4> dHeightMap)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dHeight);
    KernGenerateHeightAndImageFromHeightmap<<<gridDim,blockDim>>>(dHeight,dImage,dHeightMap);
}

}
