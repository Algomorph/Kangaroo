#include "hip/hip_runtime.h"
#include "kernel.h"

#include "CUDA_SDK/cutil_math.h"
#include <boost/math/common_factor.hpp>

using namespace std;
using namespace boost::math;

//////////////////////////////////////////////////////
// Additions to cutil_math.h
//////////////////////////////////////////////////////

inline __host__ __device__ float3 operator*(float b, uchar3 a)
{
    return make_float3(b * a.x, b * a.y, b * a.z);
}

inline __host__ __device__ float3 operator*(uchar3 a, float b)
{
    return make_float3(b * a.x, b * a.y, b * a.z);
}

//////////////////////////////////////////////////////
// Sampling
//////////////////////////////////////////////////////

// w0, w1, w2, and w3 are the four cubic B-spline basis functions
__host__ __device__
float w0(float a)
{
//    return (1.0f/6.0f)*(-a*a*a + 3.0f*a*a - 3.0f*a + 1.0f);
    return (1.0f/6.0f)*(a*(a*(-a + 3.0f) - 3.0f) + 1.0f);   // optimized
}

__host__ __device__
float w1(float a)
{
//    return (1.0f/6.0f)*(3.0f*a*a*a - 6.0f*a*a + 4.0f);
    return (1.0f/6.0f)*(a*a*(3.0f*a - 6.0f) + 4.0f);
}

__host__ __device__
float w2(float a)
{
//    return (1.0f/6.0f)*(-3.0f*a*a*a + 3.0f*a*a + 3.0f*a + 1.0f);
    return (1.0f/6.0f)*(a*(a*(-3.0f*a + 3.0f) + 3.0f) + 1.0f);
}

__host__ __device__
float w3(float a)
{
    return (1.0f/6.0f)*(a*a*a);
}

// g0 and g1 are the two amplitude functions
__device__ float g0(float a)
{
    return w0(a) + w1(a);
}

__device__ float g1(float a)
{
    return w2(a) + w3(a);
}

// h0 and h1 are the two offset functions
__device__ float h0(float a)
{
    // note +0.5 offset to compensate for CUDA linear filtering convention
    return -1.0f + w1(a) / (w0(a) + w1(a)) + 0.5f;
}

__device__ float h1(float a)
{
    return 1.0f + w3(a) / (w2(a) + w3(a)) + 0.5f;
}

// filter 4 values using cubic splines
template<typename R, typename T>
__device__
R cubicFilter(float x, T c0, T c1, T c2, T c3)
{
    R r;
    r = c0 * w0(x);
    r += c1 * w1(x);
    r += c2 * w2(x);
    r += c3 * w3(x);
    return r;
}

// Catmull-Rom interpolation

__host__ __device__
float catrom_w0(float a)
{
    //return -0.5f*a + a*a - 0.5f*a*a*a;
    return a*(-0.5f + a*(1.0f - 0.5f*a));
}

__host__ __device__
float catrom_w1(float a)
{
    //return 1.0f - 2.5f*a*a + 1.5f*a*a*a;
    return 1.0f + a*a*(-2.5f + 1.5f*a);
}

__host__ __device__
float catrom_w2(float a)
{
    //return 0.5f*a + 2.0f*a*a - 1.5f*a*a*a;
    return a*(0.5f + a*(2.0f - 1.5f*a));
}

__host__ __device__
float catrom_w3(float a)
{
    //return -0.5f*a*a + 0.5f*a*a*a;
    return a*a*(-0.5f + 0.5f*a);
}

template<typename R, typename T>
__device__
R catRomFilter(float x, T c0, T c1, T c2, T c3)
{
    R r;
    r = c0 * catrom_w0(x);
    r += c1 * catrom_w1(x);
    r += c2 * catrom_w2(x);
    r += c3 * catrom_w3(x);
    return r;
}

template<typename R, typename T>
__device__ R nearestneighbour(const T* img, int stride, float x, float y)
{
  const int xi = floor(x);
  const int yi = floor(y);
  return img[xi + stride*yi];
}

template<typename R, typename T>
__device__ R bilinear(const T* img, int stride, float x, float y)
{
  const float px = x - 0.5f;
  const float py = y - 0.5f;

//  if( 0.0 <= px && px < w-1.0 && 0.0 <= py && py < h-1.0 ) {
    const float ix = floorf(px);
    const float iy = floorf(py);
    const float fx = px - ix;
    const float fy = py - iy;
    const int idx = (int)ix + (int)iy*stride;

    return lerp(
      lerp( img[idx], img[idx+1], fx ),
      lerp( img[idx+stride], img[idx+stride+1], fx ),
      fy
    );
//  }else{
//    return nearestneighbour(img,stride,w,h,x,y);
//  }
}

template<typename R, typename T>
__device__ R bicubic(const T* img, int stride, float x, float y)
{
  const float px = x-0.5f;
  const float py = y-0.5f;

//  if( 1.0 <= px && px < w-2.0 && 1.0 <= py && py < h-2.0 ) {
    const int ix = floor(px);
    const int iy = floor(py);
    const float fx = px - ix;
    const float fy = py - iy;
    const int idx = ((int)ix) + ((int)iy)*stride;

    return cubicFilter<R,R>(
          fy,
          cubicFilter<R,T>(fx, img[idx-stride-1], img[idx-stride], img[idx-stride+1], img[idx-stride+2]),
          cubicFilter<R,T>(fx, img[idx-1], img[idx], img[idx+1], img[idx+2]),
          cubicFilter<R,T>(fx, img[idx+stride-1], img[idx+stride], img[idx+stride+1], img[idx+stride+2]),
          cubicFilter<R,T>(fx, img[idx+2*stride-1], img[idx+2*stride], img[idx+2*stride+1], img[idx+2*stride+2])
    );
//  }else{
//    return nearestneighbour(img,stride,w,h,x,y);
//  }
}

template<typename R, typename T>
__device__ R catrom(const T* img, uint stride, float x, float y)
{
  const float px = x-0.5f;
  const float py = y-0.5f;

//  if( 1.0 <= px && px < w-2.0 && 1.0 <= py && py < h-2.0 ) {
    const int ix = floor(px);
    const int iy = floor(py);
    const float fx = px - ix;
    const float fy = py - iy;
    const uint idx = ((uint)ix) + ((uint)iy)*stride;
    const uint stride2 = 2 *stride;

    return catRomFilter<R,R>(
          fy,
          catRomFilter<R,T>(fx, img[idx-stride-1], img[idx-stride], img[idx-stride+1], img[idx-stride+2]),
          catRomFilter<R,T>(fx, img[idx-1], img[idx], img[idx+1], img[idx+2]),
          catRomFilter<R,T>(fx, img[idx+stride-1], img[idx+stride], img[idx+stride+1], img[idx+stride+2]),
          catRomFilter<R,T>(fx, img[idx+stride2-1], img[idx+stride2], img[idx+stride2+1], img[idx+stride2+2])
    );
//  }else{
//    return nearestneighbour<R,T>(img,stride,x,y);
//  }
}

__global__ void  resample_kernal(
    float4* out, int ostride, int ow, int oh,
    float4* in,  int istride, int iw, int ih,
    int resample_type
) {
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int index = y*ostride + x;

    const float xf = ((x+0.5) / (float)ow) * (float)iw;
    const float yf = ((y+0.5) / (float)oh) * (float)ih;

    if( 1.5 <= xf && xf < iw-2.5 && 1.5 <= yf && yf < ih-2.5 ) {
      if( resample_type == 1 ) {
        out[index] = bilinear<float4,float4>(in,istride,xf,yf);
      }else if( resample_type == 2 ) {
        out[index] = bicubic<float4,float4>(in,istride,xf,yf);
      }else if( resample_type == 3 ) {
        out[index] = catrom<float4,float4>(in,istride,xf,yf);
      }else{
        out[index] = nearestneighbour<float4,float4>(in,istride,xf,yf);
      }
    }
}


void resample(
    float4* out, int ostride, int ow, int oh,
    float4* in,  int istride, int iw, int ih,
    int resample_type
) {
  dim3 blockdim(boost::math::gcd<unsigned>(ow,16), boost::math::gcd<unsigned>(oh,16), 1);
  dim3 griddim( ow / blockdim.x, oh / blockdim.y);
  resample_kernal<<<griddim,blockdim>>>(out,ostride,ow,oh,in,istride,iw,ih, resample_type);
}

//////////////////////////////////////////////////////
// Image warping
//////////////////////////////////////////////////////

namespace Gpu
{

//! Utility for attempting to estimate safe block/grid dimensions from working image dimensions
//! These are not necesserily optimal. Far from it.
template<typename T>
inline void InitDimFromOutputImage(dim3& blockDim, dim3& gridDim, const Image<T>& image, int blockx = 16, int blocky = 16)
{
    blockDim = dim3(boost::math::gcd<unsigned>(image.w,blockx), boost::math::gcd<unsigned>(image.h,blocky), 1);
    gridDim =  dim3( image.w / blockDim.x, image.h / blockDim.y, 1);
}

__global__ void KernCreateMatlabLookupTable(
    Image<LookupWeights> lookup, float fu, float fv, float u0, float v0, float k1, float k2
) {

}

void CreateMatlabLookupTable(
    Image<LookupWeights> lookup, float fu, float fv, float u0, float v0, float k1, float k2
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, lookup);
    KernCreateMatlabLookupTable<<<gridDim,blockDim>>>(lookup,fu,fv,u0,v0,k1,k2);
}

__global__ void KernMakeAnaglyth(Image<uchar4> anaglyth, const Image<uchar1> left, const Image<uchar1> right)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    const unsigned char leftI  = left(x,y).x;
    const unsigned char rightI = right(x,y).x;

    anaglyth(x,y) = make_uchar4(leftI, 0, rightI,255);
}

void MakeAnaglyth(Image<uchar4> anaglyth, const Image<uchar1> left, const Image<uchar1> right)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, anaglyth);
    KernMakeAnaglyth<<<gridDim,blockDim>>>(anaglyth, left, right);
}

}
