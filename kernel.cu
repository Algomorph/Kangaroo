#include "hip/hip_runtime.h"
#include "kernel.h"

#include "CUDA_SDK/cutil_math.h"
#include <boost/math/common_factor.hpp>

using namespace std;
using namespace boost::math;

//////////////////////////////////////////////////////
// Additions to cutil_math.h
//////////////////////////////////////////////////////

inline __host__ __device__ float3 operator*(float b, uchar3 a)
{
    return make_float3(b * a.x, b * a.y, b * a.z);
}

inline __host__ __device__ float3 operator*(uchar3 a, float b)
{
    return make_float3(b * a.x, b * a.y, b * a.z);
}

inline __host__ __device__ float1 operator*(float b, uchar1 a)
{
    return make_float1(b * a.x);
}

inline __host__ __device__ float1 operator*(uchar1 a, float b)
{
    return make_float1(b * a.x);
}

//////////////////////////////////////////////////////
// Sampling
//////////////////////////////////////////////////////

// w0, w1, w2, and w3 are the four cubic B-spline basis functions
__host__ __device__
float w0(float a)
{
//    return (1.0f/6.0f)*(-a*a*a + 3.0f*a*a - 3.0f*a + 1.0f);
    return (1.0f/6.0f)*(a*(a*(-a + 3.0f) - 3.0f) + 1.0f);   // optimized
}

__host__ __device__
float w1(float a)
{
//    return (1.0f/6.0f)*(3.0f*a*a*a - 6.0f*a*a + 4.0f);
    return (1.0f/6.0f)*(a*a*(3.0f*a - 6.0f) + 4.0f);
}

__host__ __device__
float w2(float a)
{
//    return (1.0f/6.0f)*(-3.0f*a*a*a + 3.0f*a*a + 3.0f*a + 1.0f);
    return (1.0f/6.0f)*(a*(a*(-3.0f*a + 3.0f) + 3.0f) + 1.0f);
}

__host__ __device__
float w3(float a)
{
    return (1.0f/6.0f)*(a*a*a);
}

// g0 and g1 are the two amplitude functions
__device__ float g0(float a)
{
    return w0(a) + w1(a);
}

__device__ float g1(float a)
{
    return w2(a) + w3(a);
}

// h0 and h1 are the two offset functions
__device__ float h0(float a)
{
    // note +0.5 offset to compensate for CUDA linear filtering convention
    return -1.0f + w1(a) / (w0(a) + w1(a)) + 0.5f;
}

__device__ float h1(float a)
{
    return 1.0f + w3(a) / (w2(a) + w3(a)) + 0.5f;
}

// filter 4 values using cubic splines
template<typename R, typename T>
__device__
R cubicFilter(float x, T c0, T c1, T c2, T c3)
{
    R r;
    r = c0 * w0(x);
    r += c1 * w1(x);
    r += c2 * w2(x);
    r += c3 * w3(x);
    return r;
}

// Catmull-Rom interpolation

__host__ __device__
float catrom_w0(float a)
{
    //return -0.5f*a + a*a - 0.5f*a*a*a;
    return a*(-0.5f + a*(1.0f - 0.5f*a));
}

__host__ __device__
float catrom_w1(float a)
{
    //return 1.0f - 2.5f*a*a + 1.5f*a*a*a;
    return 1.0f + a*a*(-2.5f + 1.5f*a);
}

__host__ __device__
float catrom_w2(float a)
{
    //return 0.5f*a + 2.0f*a*a - 1.5f*a*a*a;
    return a*(0.5f + a*(2.0f - 1.5f*a));
}

__host__ __device__
float catrom_w3(float a)
{
    //return -0.5f*a*a + 0.5f*a*a*a;
    return a*a*(-0.5f + 0.5f*a);
}

template<typename R, typename T>
__device__
R catRomFilter(float x, T c0, T c1, T c2, T c3)
{
    R r;
    r = c0 * catrom_w0(x);
    r += c1 * catrom_w1(x);
    r += c2 * catrom_w2(x);
    r += c3 * catrom_w3(x);
    return r;
}

template<typename R, typename T>
__device__ R nearestneighbour(const T* img, int stride, float x, float y)
{
  const int xi = floor(x);
  const int yi = floor(y);
  return img[xi + stride*yi];
}

template<typename R, typename T>
__device__ R bilinear(const T* img, int stride, float x, float y)
{
  const float px = x - 0.5f;
  const float py = y - 0.5f;

//  if( 0.0 <= px && px < w-1.0 && 0.0 <= py && py < h-1.0 ) {
    const float ix = floorf(px);
    const float iy = floorf(py);
    const float fx = px - ix;
    const float fy = py - iy;
    const int idx = (int)ix + (int)iy*stride;

    return lerp(
      lerp( img[idx], img[idx+1], fx ),
      lerp( img[idx+stride], img[idx+stride+1], fx ),
      fy
    );
//  }else{
//    return nearestneighbour(img,stride,w,h,x,y);
//  }
}

template<typename R, typename T>
__device__ R bicubic(const T* img, int stride, float x, float y)
{
  const float px = x-0.5f;
  const float py = y-0.5f;

//  if( 1.0 <= px && px < w-2.0 && 1.0 <= py && py < h-2.0 ) {
    const int ix = floor(px);
    const int iy = floor(py);
    const float fx = px - ix;
    const float fy = py - iy;
    const int idx = ((int)ix) + ((int)iy)*stride;

    return cubicFilter<R,R>(
          fy,
          cubicFilter<R,T>(fx, img[idx-stride-1], img[idx-stride], img[idx-stride+1], img[idx-stride+2]),
          cubicFilter<R,T>(fx, img[idx-1], img[idx], img[idx+1], img[idx+2]),
          cubicFilter<R,T>(fx, img[idx+stride-1], img[idx+stride], img[idx+stride+1], img[idx+stride+2]),
          cubicFilter<R,T>(fx, img[idx+2*stride-1], img[idx+2*stride], img[idx+2*stride+1], img[idx+2*stride+2])
    );
//  }else{
//    return nearestneighbour(img,stride,w,h,x,y);
//  }
}

template<typename R, typename T>
__device__ R catrom(const T* img, uint stride, float x, float y)
{
  const float px = x-0.5f;
  const float py = y-0.5f;

//  if( 1.0 <= px && px < w-2.0 && 1.0 <= py && py < h-2.0 ) {
    const int ix = floor(px);
    const int iy = floor(py);
    const float fx = px - ix;
    const float fy = py - iy;
    const uint idx = ((uint)ix) + ((uint)iy)*stride;
    const uint stride2 = 2 *stride;

    return catRomFilter<R,R>(
          fy,
          catRomFilter<R,T>(fx, img[idx-stride-1], img[idx-stride], img[idx-stride+1], img[idx-stride+2]),
          catRomFilter<R,T>(fx, img[idx-1], img[idx], img[idx+1], img[idx+2]),
          catRomFilter<R,T>(fx, img[idx+stride-1], img[idx+stride], img[idx+stride+1], img[idx+stride+2]),
          catRomFilter<R,T>(fx, img[idx+stride2-1], img[idx+stride2], img[idx+stride2+1], img[idx+stride2+2])
    );
//  }else{
//    return nearestneighbour<R,T>(img,stride,x,y);
//  }
}

__global__ void  resample_kernal(
    float4* out, int ostride, int ow, int oh,
    float4* in,  int istride, int iw, int ih,
    int resample_type
) {
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int index = y*ostride + x;

    const float xf = ((x+0.5) / (float)ow) * (float)iw;
    const float yf = ((y+0.5) / (float)oh) * (float)ih;

    if( 1.5 <= xf && xf < iw-2.5 && 1.5 <= yf && yf < ih-2.5 ) {
      if( resample_type == 1 ) {
        out[index] = bilinear<float4,float4>(in,istride,xf,yf);
      }else if( resample_type == 2 ) {
        out[index] = bicubic<float4,float4>(in,istride,xf,yf);
      }else if( resample_type == 3 ) {
        out[index] = catrom<float4,float4>(in,istride,xf,yf);
      }else{
        out[index] = nearestneighbour<float4,float4>(in,istride,xf,yf);
      }
    }
}


void resample(
    float4* out, int ostride, int ow, int oh,
    float4* in,  int istride, int iw, int ih,
    int resample_type
) {
  dim3 blockdim(boost::math::gcd<unsigned>(ow,16), boost::math::gcd<unsigned>(oh,16), 1);
  dim3 griddim( ow / blockdim.x, oh / blockdim.y);
  resample_kernal<<<griddim,blockdim>>>(out,ostride,ow,oh,in,istride,iw,ih, resample_type);
}

//////////////////////////////////////////////////////
// Image warping
//////////////////////////////////////////////////////

namespace Gpu
{

//! Utility for attempting to estimate safe block/grid dimensions from working image dimensions
//! These are not necesserily optimal. Far from it.
template<typename T>
inline void InitDimFromOutputImage(dim3& blockDim, dim3& gridDim, const Image<T>& image, int blockx = 16, int blocky = 16)
{
    blockDim = dim3(boost::math::gcd<unsigned>(image.w,blockx), boost::math::gcd<unsigned>(image.h,blocky), 1);
    gridDim =  dim3( image.w / blockDim.x, image.h / blockDim.y, 1);
}

//////////////////////////////////////////////////////
// Create Matlab Lookup table
//////////////////////////////////////////////////////

__global__ void KernCreateMatlabLookupTable(
    Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2
) {
    const uint u = blockIdx.x*blockDim.x + threadIdx.x;
    const uint v = blockIdx.y*blockDim.y + threadIdx.y;

    const float pnu = (u-u0) / fu;
    const float pnv = (v-v0) / fv;
    const float r = sqrt(pnu*pnu + pnv*pnv);
    const float rr = r*r;
    const float rf = 1 + k1*rr + k2*rr*rr; // + k3*rr*rr*rr;

    lookup(u,v) = make_float2(
        (pnu*rf /*+ 2*p1*pn.x*pn.y + p2*(rr + 2*pn.x*pn.x)*/) * fu + u0,
        (pnv*rf /*+ p1*(rr + 2*pn.y*pn.y) + 2*p2*pn.x*pn.y*/) * fv + v0
    );
}

void CreateMatlabLookupTable(
    Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, lookup);
    KernCreateMatlabLookupTable<<<gridDim,blockDim>>>(lookup,fu,fv,u0,v0,k1,k2);
}

//////////////////////////////////////////////////////
// Create Matlab Lookup table applying homography
//////////////////////////////////////////////////////

__global__ void KernCreateMatlabLookupTable(
    Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2, Array<float,9> H_on
) {
    const uint x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint y = blockIdx.y*blockDim.y + threadIdx.y;

    // Apply homography H_on, moving New image to Original
    const float hdiv = H_on[6] * x + H_on[7] * y + H_on[8];
    const float u = (H_on[0] * x + H_on[1] * y + H_on[2]) / hdiv;
    const float v = (H_on[3] * x + H_on[4] * y + H_on[5]) / hdiv;

    // Apply distortion to achieve true image coordinates
    const float pnu = (u-u0) / fu;
    const float pnv = (v-v0) / fv;
    const float r = sqrt(pnu*pnu + pnv*pnv);
    const float rr = r*r;
    const float rf = 1 + k1*rr + k2*rr*rr; // + k3*rr*rr*rr;

    float2 pos = make_float2(
        (pnu*rf /*+ 2*p1*pn.x*pn.y + p2*(rr + 2*pn.x*pn.x)*/) * fu + u0,
        (pnv*rf /*+ p1*(rr + 2*pn.y*pn.y) + 2*p2*pn.x*pn.y*/) * fv + v0
    );

    // Clamp to image bounds
    pos.x = max(pos.x, 1.0f);
    pos.y = max(pos.y, 1.0f);
    pos.x = min(pos.x, lookup.w-2.0f);
    pos.y = min(pos.y, lookup.h-2.0f);

    lookup(x,y) = pos;
}

void CreateMatlabLookupTable(
        Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2, Array<float,9> H_no
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, lookup);
    KernCreateMatlabLookupTable<<<gridDim,blockDim>>>(lookup,fu,fv,u0,v0,k1,k2,H_no);
}

//////////////////////////////////////////////////////
// Warp image using lookup table
//////////////////////////////////////////////////////

__global__ void KernWarp(
    Image<unsigned char> out, const Image<unsigned char> in, const Image<float2> lookup
) {
    const uint x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint y = blockIdx.y*blockDim.y + threadIdx.y;

    const float2 lu = lookup(x,y);
    out(x,y) = bicubic<float,unsigned char>((unsigned char*)in.ptr, in.stride, lu.x, lu.y);
}

void Warp(
    Image<unsigned char> out, const Image<unsigned char> in, const Image<float2> lookup
) {
    assert(out.w <= lookup.w && out.h <= lookup.h);
    assert(out.w <= in.w && out.h <= in.w);

    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, out);
    KernWarp<<<gridDim,blockDim>>>(out, in, lookup);

}

//////////////////////////////////////////////////////
// Patch Scores
//////////////////////////////////////////////////////

template<typename T, int size>
__device__ inline
float Sum(
    Image<T> img, int x, int y
) {
    float sum = 0;
    for(int r=-size; r <=size; ++r ) {
        for(int c=-size; c <=size; ++c ) {
            sum += img.GetWithClampedRange(x+c,y+r);
        }
    }
    return sum;
}

// Mean Absolute Difference
template<typename T, int rad>
__device__ inline
float MADScore(
    Image<T> img1, int x1, int y1,
    Image<T> img2, int x2, int y2
) {
    const int w = 2*rad+1;
    float sum_abs_diff = 0;

    for(int r=-rad; r <=rad; ++r ) {
        for(int c=-rad; c <=rad; ++c ) {
            float i1 = img1.GetWithClampedRange(x1+c,y1+r);
            float i2 = img2.GetWithClampedRange(x2+c,y2+r);
            sum_abs_diff += abs(i1 - i2);
        }
    }

    return sum_abs_diff / (w*w);
}

// Mean Normalised Difference
template<typename T, int rad>
__device__ inline
float MNDScore(
    Image<T> img1, int x1, int y1,
    Image<T> img2, int x2, int y2
) {
    const int w = 2*rad+1;
    const float m1 = Sum<T,rad>(img1,x1,y1) / (w*w);
    const float m2 = Sum<T,rad>(img2,x2,y2) / (w*w);

    float sum_abs_diff = 0;

    for(int r=-rad; r <=rad; ++r ) {
        for(int c=-rad; c <=rad; ++c ) {
            float i1 = img1.GetWithClampedRange(x1+c,y1+r) - m1;
//            float i2 = img1.GetWithClampedRange(x2+c,y2+r) - m1;
            float i2 = img2.GetWithClampedRange(x2+c,y2+r) - m2;
            sum_abs_diff += abs(i1 - i2);
        }
    }

    return sum_abs_diff;
}

//////////////////////////////////////////////////////
// Scanline rectified dense stereo
//////////////////////////////////////////////////////

template<typename T, unsigned int rad>
__global__ void KernDenseStereo(
    Image<float> dDisp, Image<T> dCamLeft, Image<T> dCamRight, int maxDisp
) {
    const uint x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint y = blockIdx.y*blockDim.y + threadIdx.y;

    // Search for best matching pixel
    int bestDisp = 0;
    float bestScore = 1E10;

    for(int c = 0; c < maxDisp; ++c ) {
        const float score = MNDScore<unsigned char,rad>(dCamLeft, x,y, dCamRight, x-c, y);
        if(score < bestScore) {
            bestScore = score;
            bestDisp = c;
        }
    }

    dDisp(x,y) = bestDisp; //(float)bestDisp / (float)maxDisp;
}

void DenseStereo(
    Image<float> dDisp, Image<unsigned char> dCamLeft, Image<unsigned char> dCamRight, int maxDisp
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dDisp);
    KernDenseStereo<unsigned char,3><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight,maxDisp);
}

//////////////////////////////////////////////////////
// Scanline rectified dense stereo sub-pixel refinement
//////////////////////////////////////////////////////

template<typename T, unsigned int rad>
__global__ void KernDenseStereoSubpixelRefine(
    Image<float> dDisp, Image<T> dCamLeft, Image<T> dCamRight, int maxDisp
) {
    const uint x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint y = blockIdx.y*blockDim.y + threadIdx.y;

    const int bestDisp = dDisp(x,y);

    // Fit parabola to neighbours
    const float d1 = bestDisp+1;
    const float d2 = bestDisp;
    const float d3 = bestDisp-1;
    const float s1 = MNDScore<unsigned char,rad>(dCamLeft, x,y, dCamRight, x-d1,y);
    const float s2 = MNDScore<unsigned char,rad>(dCamLeft, x,y, dCamRight, x-d2,y);
    const float s3 = MNDScore<unsigned char,rad>(dCamLeft, x,y, dCamRight, x-d3,y);

    // Cooefficients of parabola through (d1,s1),(d2,s2),(d3,s3)
    const float denom = (d1 - d2)*(d1 - d3)*(d2 - d3);
    const float A = (d3 * (s2 - s1) + d2 * (s1 - s3) + d1 * (s3 - s2)) / denom;
    const float B = (d3*d3 * (s1 - s2) + d2*d2 * (s3 - s1) + d1*d1 * (s2 - s3)) / denom;
//    const float C = (x2 * x3 * (x2 - x3) * y1 + x3 * x1 * (x3 - x1) * y2 + x1 * x2 * (x1 - x2) * y3) / denom;

    // Minima of parabola
    const float newDisp = -B / (2*A);

    // Check that minima is sensible. Maybe we don't really need to do this.
    if( d3 < newDisp && newDisp < d1 ) {
        dDisp(x,y) = newDisp / (float)maxDisp;
    }else{
        dDisp(x,y) = bestDisp / (float)maxDisp;
    }
}

void DenseStereoSubpixelRefine(
    Image<float> dDisp, Image<unsigned char> dCamLeft, Image<unsigned char> dCamRight, int maxDisp
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dDisp);
    KernDenseStereoSubpixelRefine<unsigned char,3><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight,maxDisp);
}

//////////////////////////////////////////////////////
// Quick and dirty Bilateral filer
//////////////////////////////////////////////////////

__global__ void KernBilateralFilter(
    Image<float> dOut, Image<float> dIn, float gs, float gr, int size
) {
    const uint x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint y = blockIdx.y*blockDim.y + threadIdx.y;

    const float p = dIn(x,y);
    float sum = 0;
    float sumw = 0;

    for(int r = -size; r <= size; ++r ) {
        for(int c = -size; c <= size; ++c ) {
            const float q = dIn.GetWithClampedRange(x+c, y+r);
            const float sd2 = r*r + c*c;
            const float id = p-q;
            const float id2 = id*id;
            const float sw = __expf(-(sd2) / (2 * gs * gs));
            const float iw = __expf(-(id2) / (2 * gr * gr));
            const float w = sw*iw;
            sumw += w;
            sum += w * q;
//            sumw += 1;
//            sum += q;
        }
    }

    dOut(x,y) = sum / sumw;
}

void BilateralFilter(
    Image<float> dOut, Image<float> dIn, float gs, float gr, uint size
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dOut);
    KernBilateralFilter<<<gridDim,blockDim>>>(dOut, dIn, gs, gr, size);
}

//////////////////////////////////////////////////////
// Anaglyph: Join left / right images into anagly stereo
//////////////////////////////////////////////////////

__global__ void KernMakeAnaglyth(Image<uchar4> anaglyth, const Image<unsigned char> left, const Image<unsigned char> right)
{
    const uint x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint y = blockIdx.y*blockDim.y + threadIdx.y;

    anaglyth(x,y) = make_uchar4(left(x,y), 0, right(x,y),255);
}

void MakeAnaglyth(Image<uchar4> anaglyth, const Image<unsigned char> left, const Image<unsigned char> right)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, anaglyth);
    KernMakeAnaglyth<<<gridDim,blockDim>>>(anaglyth, left, right);
}

}
