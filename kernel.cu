#include "hip/hip_runtime.h"
#include "kernel.h"

#include "CUDA_SDK/cutil_math.h"
#include <boost/math/common_factor.hpp>

using namespace std;
using namespace boost::math;

//////////////////////////////////////////////////////
// Additions to cutil_math.h
//////////////////////////////////////////////////////

inline __host__ __device__ float3 operator*(float b, uchar3 a)
{
    return make_float3(b * a.x, b * a.y, b * a.z);
}

inline __host__ __device__ float3 operator*(uchar3 a, float b)
{
    return make_float3(b * a.x, b * a.y, b * a.z);
}

inline __host__ __device__ float1 operator*(float b, uchar1 a)
{
    return make_float1(b * a.x);
}

inline __host__ __device__ float1 operator*(uchar1 a, float b)
{
    return make_float1(b * a.x);
}

//////////////////////////////////////////////////////
// Sampling
//////////////////////////////////////////////////////

// w0, w1, w2, and w3 are the four cubic B-spline basis functions
__host__ __device__
float w0(float a)
{
//    return (1.0f/6.0f)*(-a*a*a + 3.0f*a*a - 3.0f*a + 1.0f);
    return (1.0f/6.0f)*(a*(a*(-a + 3.0f) - 3.0f) + 1.0f);   // optimized
}

__host__ __device__
float w1(float a)
{
//    return (1.0f/6.0f)*(3.0f*a*a*a - 6.0f*a*a + 4.0f);
    return (1.0f/6.0f)*(a*a*(3.0f*a - 6.0f) + 4.0f);
}

__host__ __device__
float w2(float a)
{
//    return (1.0f/6.0f)*(-3.0f*a*a*a + 3.0f*a*a + 3.0f*a + 1.0f);
    return (1.0f/6.0f)*(a*(a*(-3.0f*a + 3.0f) + 3.0f) + 1.0f);
}

__host__ __device__
float w3(float a)
{
    return (1.0f/6.0f)*(a*a*a);
}

// g0 and g1 are the two amplitude functions
__device__ float g0(float a)
{
    return w0(a) + w1(a);
}

__device__ float g1(float a)
{
    return w2(a) + w3(a);
}

// h0 and h1 are the two offset functions
__device__ float h0(float a)
{
    // note +0.5 offset to compensate for CUDA linear filtering convention
    return -1.0f + w1(a) / (w0(a) + w1(a)) + 0.5f;
}

__device__ float h1(float a)
{
    return 1.0f + w3(a) / (w2(a) + w3(a)) + 0.5f;
}

// filter 4 values using cubic splines
template<typename R, typename T>
__device__
R cubicFilter(float x, T c0, T c1, T c2, T c3)
{
    R r;
    r = c0 * w0(x);
    r += c1 * w1(x);
    r += c2 * w2(x);
    r += c3 * w3(x);
    return r;
}

// Catmull-Rom interpolation

__host__ __device__
float catrom_w0(float a)
{
    //return -0.5f*a + a*a - 0.5f*a*a*a;
    return a*(-0.5f + a*(1.0f - 0.5f*a));
}

__host__ __device__
float catrom_w1(float a)
{
    //return 1.0f - 2.5f*a*a + 1.5f*a*a*a;
    return 1.0f + a*a*(-2.5f + 1.5f*a);
}

__host__ __device__
float catrom_w2(float a)
{
    //return 0.5f*a + 2.0f*a*a - 1.5f*a*a*a;
    return a*(0.5f + a*(2.0f - 1.5f*a));
}

__host__ __device__
float catrom_w3(float a)
{
    //return -0.5f*a*a + 0.5f*a*a*a;
    return a*a*(-0.5f + 0.5f*a);
}

template<typename R, typename T>
__device__
R catRomFilter(float x, T c0, T c1, T c2, T c3)
{
    R r;
    r = c0 * catrom_w0(x);
    r += c1 * catrom_w1(x);
    r += c2 * catrom_w2(x);
    r += c3 * catrom_w3(x);
    return r;
}

template<typename R, typename T>
__device__ R nearestneighbour(const T* img, int stride, float x, float y)
{
  const int xi = floor(x);
  const int yi = floor(y);
  return img[xi + stride*yi];
}

template<typename R, typename T>
__device__ R bilinear(const T* img, int stride, float x, float y)
{
  const float px = x - 0.5f;
  const float py = y - 0.5f;

//  if( 0.0 <= px && px < w-1.0 && 0.0 <= py && py < h-1.0 ) {
    const float ix = floorf(px);
    const float iy = floorf(py);
    const float fx = px - ix;
    const float fy = py - iy;
    const int idx = (int)ix + (int)iy*stride;

    return lerp(
      lerp( img[idx], img[idx+1], fx ),
      lerp( img[idx+stride], img[idx+stride+1], fx ),
      fy
    );
//  }else{
//    return nearestneighbour(img,stride,w,h,x,y);
//  }
}

template<typename R, typename T>
__device__ R bicubic(const T* img, int stride, float x, float y)
{
  const float px = x-0.5f;
  const float py = y-0.5f;

//  if( 1.0 <= px && px < w-2.0 && 1.0 <= py && py < h-2.0 ) {
    const int ix = floor(px);
    const int iy = floor(py);
    const float fx = px - ix;
    const float fy = py - iy;
    const int idx = ((int)ix) + ((int)iy)*stride;

    return cubicFilter<R,R>(
          fy,
          cubicFilter<R,T>(fx, img[idx-stride-1], img[idx-stride], img[idx-stride+1], img[idx-stride+2]),
          cubicFilter<R,T>(fx, img[idx-1], img[idx], img[idx+1], img[idx+2]),
          cubicFilter<R,T>(fx, img[idx+stride-1], img[idx+stride], img[idx+stride+1], img[idx+stride+2]),
          cubicFilter<R,T>(fx, img[idx+2*stride-1], img[idx+2*stride], img[idx+2*stride+1], img[idx+2*stride+2])
    );
//  }else{
//    return nearestneighbour(img,stride,w,h,x,y);
//  }
}

template<typename R, typename T>
__device__ R catrom(const T* img, uint stride, float x, float y)
{
  const float px = x-0.5f;
  const float py = y-0.5f;

//  if( 1.0 <= px && px < w-2.0 && 1.0 <= py && py < h-2.0 ) {
    const int ix = floor(px);
    const int iy = floor(py);
    const float fx = px - ix;
    const float fy = py - iy;
    const uint idx = ((uint)ix) + ((uint)iy)*stride;
    const uint stride2 = 2 *stride;

    return catRomFilter<R,R>(
          fy,
          catRomFilter<R,T>(fx, img[idx-stride-1], img[idx-stride], img[idx-stride+1], img[idx-stride+2]),
          catRomFilter<R,T>(fx, img[idx-1], img[idx], img[idx+1], img[idx+2]),
          catRomFilter<R,T>(fx, img[idx+stride-1], img[idx+stride], img[idx+stride+1], img[idx+stride+2]),
          catRomFilter<R,T>(fx, img[idx+stride2-1], img[idx+stride2], img[idx+stride2+1], img[idx+stride2+2])
    );
//  }else{
//    return nearestneighbour<R,T>(img,stride,x,y);
//  }
}

__global__ void  resample_kernal(
    float4* out, int ostride, int ow, int oh,
    float4* in,  int istride, int iw, int ih,
    int resample_type
) {
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int index = y*ostride + x;

    const float xf = ((x+0.5) / (float)ow) * (float)iw;
    const float yf = ((y+0.5) / (float)oh) * (float)ih;

    if( 1.5 <= xf && xf < iw-2.5 && 1.5 <= yf && yf < ih-2.5 ) {
      if( resample_type == 1 ) {
        out[index] = bilinear<float4,float4>(in,istride,xf,yf);
      }else if( resample_type == 2 ) {
        out[index] = bicubic<float4,float4>(in,istride,xf,yf);
      }else if( resample_type == 3 ) {
        out[index] = catrom<float4,float4>(in,istride,xf,yf);
      }else{
        out[index] = nearestneighbour<float4,float4>(in,istride,xf,yf);
      }
    }
}


void resample(
    float4* out, int ostride, int ow, int oh,
    float4* in,  int istride, int iw, int ih,
    int resample_type
) {
  dim3 blockdim(boost::math::gcd<unsigned>(ow,16), boost::math::gcd<unsigned>(oh,16), 1);
  dim3 griddim( ow / blockdim.x, oh / blockdim.y);
  resample_kernal<<<griddim,blockdim>>>(out,ostride,ow,oh,in,istride,iw,ih, resample_type);
}

//////////////////////////////////////////////////////
// Image warping
//////////////////////////////////////////////////////

namespace Gpu
{

//! Utility for attempting to estimate safe block/grid dimensions from working image dimensions
//! These are not necesserily optimal. Far from it.
template<typename T>
inline void InitDimFromOutputImage(dim3& blockDim, dim3& gridDim, const Image<T>& image, int blockx = 16, int blocky = 16)
{
    blockDim = dim3(boost::math::gcd<unsigned>(image.w,blockx), boost::math::gcd<unsigned>(image.h,blocky), 1);
    gridDim =  dim3( image.w / blockDim.x, image.h / blockDim.y, 1);
}

//////////////////////////////////////////////////////
// Create Matlab Lookup table
//////////////////////////////////////////////////////

__global__ void KernCreateMatlabLookupTable(
    Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2
) {
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;

    const float pnu = (u-u0) / fu;
    const float pnv = (v-v0) / fv;
    const float r = sqrt(pnu*pnu + pnv*pnv);
    const float rr = r*r;
    const float rf = 1 + k1*rr + k2*rr*rr; // + k3*rr*rr*rr;

    lookup(u,v) = make_float2(
        (pnu*rf /*+ 2*p1*pn.x*pn.y + p2*(rr + 2*pn.x*pn.x)*/) * fu + u0,
        (pnv*rf /*+ p1*(rr + 2*pn.y*pn.y) + 2*p2*pn.x*pn.y*/) * fv + v0
    );
}

void CreateMatlabLookupTable(
    Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, lookup);
    KernCreateMatlabLookupTable<<<gridDim,blockDim>>>(lookup,fu,fv,u0,v0,k1,k2);
}

//////////////////////////////////////////////////////
// Create Matlab Lookup table applying homography
//////////////////////////////////////////////////////

__global__ void KernCreateMatlabLookupTable(
    Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2, Array<float,9> H_on
) {
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    // Apply homography H_on, moving New image to Original
    const float hdiv = H_on[6] * x + H_on[7] * y + H_on[8];
    const float u = (H_on[0] * x + H_on[1] * y + H_on[2]) / hdiv;
    const float v = (H_on[3] * x + H_on[4] * y + H_on[5]) / hdiv;

    // Apply distortion to achieve true image coordinates
    const float pnu = (u-u0) / fu;
    const float pnv = (v-v0) / fv;
    const float r = sqrt(pnu*pnu + pnv*pnv);
    const float rr = r*r;
    const float rf = 1 + k1*rr + k2*rr*rr; // + k3*rr*rr*rr;

    float2 pos = make_float2(
        (pnu*rf /*+ 2*p1*pn.x*pn.y + p2*(rr + 2*pn.x*pn.x)*/) * fu + u0,
        (pnv*rf /*+ p1*(rr + 2*pn.y*pn.y) + 2*p2*pn.x*pn.y*/) * fv + v0
    );

    // Clamp to image bounds
    pos.x = max(pos.x, 1.0f);
    pos.y = max(pos.y, 1.0f);
    pos.x = min(pos.x, lookup.w-2.0f);
    pos.y = min(pos.y, lookup.h-2.0f);

    lookup(x,y) = pos;
}

void CreateMatlabLookupTable(
        Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2, Array<float,9> H_no
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, lookup);
    KernCreateMatlabLookupTable<<<gridDim,blockDim>>>(lookup,fu,fv,u0,v0,k1,k2,H_no);
}

//////////////////////////////////////////////////////
// Warp image using lookup table
//////////////////////////////////////////////////////

__global__ void KernWarp(
    Image<uchar1> out, const Image<uchar1> in, const Image<float2> lookup
) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    const float2 lu = lookup(x,y);
    out(x,y).x = bicubic<float,unsigned char>((unsigned char*)in.ptr, in.stride, lu.x, lu.y);
}

void Warp(
    Image<uchar1> out, const Image<uchar1> in, const Image<float2> lookup
) {
    assert(out.w <= lookup.w && out.h <= lookup.h);
    assert(out.w <= in.w && out.h <= in.w);

    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, out);
    KernWarp<<<gridDim,blockDim>>>(out, in, lookup);

}

//////////////////////////////////////////////////////
// Anaglyph: Join left / right images into anagly stereo
//////////////////////////////////////////////////////

__global__ void KernMakeAnaglyth(Image<uchar4> anaglyth, const Image<uchar1> left, const Image<uchar1> right)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    const unsigned char leftI  = left(x,y).x;
    const unsigned char rightI = right(x,y).x;

    anaglyth(x,y) = make_uchar4(leftI, 0, rightI,255);
}

void MakeAnaglyth(Image<uchar4> anaglyth, const Image<uchar1> left, const Image<uchar1> right)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, anaglyth);
    KernMakeAnaglyth<<<gridDim,blockDim>>>(anaglyth, left, right);
}

}
